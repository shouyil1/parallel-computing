#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 16
#define TILE_SIZE (N / nStreams) 

// Kernel to perform matrix multiplication on a tile
__global__ void matMulKernel(int *A, int *B, int *C, int A_offset, int C_offset) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + A_offset;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int Cvalue = 0;
    for (int k = 0; k < N; ++k) {
        Cvalue += A[row * N + k] * B[k * N + col];
    }
    C[(row-A_offset) * N + col + C_offset] = Cvalue;
}


// Main program
int main() {
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(int);
    // int nStreams = 4;
    // int TILE_SIZE = (N / nStreams);

    // Allocate host memory
    hipHostMalloc((void**)&A, size);
    hipHostMalloc((void**)&B, size);
    hipHostMalloc((void**)&C, size);

    // Initialize matrices
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = i;
            B[i * N + j] = j;
        }
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy B to device synchronously as it is needed in full for each computation
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Create events and streams
    hipEvent_t startEvent, stopEvent, dummyEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&dummyEvent);

    hipStream_t streams[nStreams];
    for (int i = 0; i < nStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Start recording
    hipEventRecord(startEvent, 0);

    // Loop over streams
    for (int i = 0; i < nStreams; ++i) {
        // Calculate offsets
        int A_offset = i * TILE_SIZE;
        int C_offset = i * TILE_SIZE * N;

        // Asynchronously copy a tile of A to the device
        hipMemcpyAsync(&d_A[A_offset * N], &A[A_offset * N], N * N * sizeof(int) / nStreams, hipMemcpyHostToDevice, streams[i]);

        // Configure grid and block dimensions
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid((64 / nStreams), 64);

        // Launch the kernel on a stream
        matMulKernel<<<dimGrid, dimBlock, 0, streams[i]>>>(d_A, d_B, d_C, A_offset, C_offset);

        // Check for any errors in kernel launch
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(err));
        }

        // Asynchronously copy a tile of C back to the host
        hipMemcpyAsync(&C[C_offset], &d_C[C_offset], N * N * sizeof(int) / nStreams, hipMemcpyDeviceToHost, streams[i]);
    }

    // Stop recording
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    printf("Time for computation and memory operations: %f ms\n", milliseconds);

    // Print the value of C[451][451]
    printf("Value of C[451][451]: %d\n", C[451 * N + 451]);

    // Cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(dummyEvent);
    for (int i = 0; i < nStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
